// #######################################################
//
// Exercício de multiplicação de matrizes em CUDA
// Disciplina: OPRP001 - Programação Paralela
// Prof.: Mauricio Pillon
// Dupla: Beatriz e Geremias
//
// #######################################################
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// Matriz Quadrada (nro_linhas = nro_colunas)
#define N 4 // Número de linhas
            // Número de colunas

// GPU: Multiplicação das matrizes (a) e (b), resultado em (c) - Tarefa para os alunos:
__global__ void matMult (int *da, int *db, int *dc) {

    /* Explicação
    Temos uma matriz 4x4, comm quantidade de threads enviadas de dimBlock * dimThreads (1 * 1 * 4 * 4 = 16)
      Pode possuir uma formatação diferente, mas, para esse caso, a multiplicação entre ambas precisa resultar em 16
      Na declaração do dimBlock e dimThreads, deixamos comentada uma possibilidade com formatação, para caso N = 4, com (2,2) (2,2)
      Mas repare que para essa formatação o seu uso é mais restrito, pois N precisa ser um inteiro com raíz quadrada exata! 
    Portanto, cada thread executa o cálculo de cada uma das células, sendo identificado por ser id de i e j para ser efetuado
    Para efetuar a multiplicação, é utilizado a ideia do for truncado, que vai de 0 ao tam da matriz (N)
    Tendo nele o padrão de que: matC[i][j] += matA[i][k] * matB[k][j]
    Obs: na função dirtyMem(() temos a garantia que não há lixo de memória nos elementos da matriz dc
    */

    int i = blockIdx.x * blockDim.x + threadIdx.x; //identifica de linha
    int j = blockIdx.y * blockDim.y + threadIdx.y; //identifica de coluna

    //For para efetuar o cálculo de multiplicação, individual a cada thread.
    for(int k = 0; k < N; k++){
      dc[i*N+j] += (da[i*N+k] * db[k*N+j]);
    }

    //Print para teste
   /* printf ("[%d][%d]=%d\t(x)\t%d\t%d\t%d\t(y)\t%d\t%d\t%d\n", \\
            i,j,(i*N+j), threadIdx.x, blockIdx.x, blockDim.x, \\
            threadIdx.y, blockIdx.y, blockDim.y);
  
    dc[i*N+j] = da[i*N+j] + db[i*N+j];*/
}

// GPU: Imprime índices na matriz
__global__ void printIndex (void) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int j = blockIdx.y * blockDim.y + threadIdx.y;

   printf ("[%d][%d]=%d\t(x)\t%d\t%d\t%d\t(y)\t%d\t%d\t%d\n",i,j,(i*N+j), threadIdx.x, blockIdx.x, blockDim.x,threadIdx.y, blockIdx.y, blockDim.y);
}


// GPU: Inicializa os vetores (a), (b) e (c) na Memória Global
__global__ void dirtyMem (int *da, int *db, int *dc) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   da[i] = -1;
   db[i] = -2;
   dc[i] = 0;
}

// CPU: Inicializa os vetores (a) e (b)
__host__ void initvet(int *host_a, int *host_b) {
  for (int i=0; i < N; i++) {
    for (int j=0; j < N; j++) {
       host_b[i*N+j] = (i+j)+((N-1)*i);
       host_a[i*N+j] = (N*N)-host_b[i*N+j];
    }
  }
}

// CPU: Imprime matriz
__host__ void printMat (int *mat){

	for (int j =0; j < N; j++)
	printf("\t(%d)", j);
	printf("\n");
	for (int i=0; i < N; i++) {
		printf("(%d)", i);
		for (int j=0; j < N; j++){
			printf("\t%d", mat[i*N+j]);
		}
		printf("\n");
	}
}


// CPU: função principal
int main(int argc, char const *argv[]) {
  int *a, *b, *c;
  int *dev_a, *dev_b, *dev_c;
  int size;

  // Alocação de matriz quadrada
  size = N * N * sizeof(int);

  // Alocação de memória no host
  hipHostMalloc((void **) &a, size, hipHostMallocDefault);
  hipHostMalloc((void **) &b, size, hipHostMallocDefault);
  hipHostMalloc((void **) &c, size, hipHostMallocDefault);

  // Alocação de memória na GPU para os vetores (a,b e c)
  hipMalloc ((void **) &dev_a, size);
  hipMalloc ((void **) &dev_b, size);
  hipMalloc ((void **) &dev_c, size);


  // Atribui valores iniciais aos vetores em GPU
  dirtyMem<<<N, N>>>(dev_a, dev_b, dev_c);

  // Cópia GPU para CPU
  hipMemcpy (a, dev_a, size, hipMemcpyDeviceToHost);
  hipMemcpy (b, dev_b, size, hipMemcpyDeviceToHost);
  hipMemcpy (c, dev_c, size, hipMemcpyDeviceToHost);

  // Impressão na tela dos valores dos vetores
  printf ("\t ### Valores Inicializados na GPU ###\n");
  printf ("\t ### Matriz (a) ### \n");
  printMat(a);
  printf ("\t ### Matriz (b) ### \n");
  printMat(b);
  printf ("\t ### Matriz (c) ### \n");
  printMat(c);

  // Inicialização dos vetores (a) e (b) no host
  initvet(a,b);

  // Cópia dos vetores gerados em CPU p/ memória da GPU
  hipMemcpy (dev_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy (dev_b, b, size, hipMemcpyHostToDevice);

  // Número de blocos e threads p/ dimensões (x,y) - formatação 1:
  dim3 dimBlock (1, 1);
  dim3 dimThreads(N, N);

  // Número de blocos e threads p/ dimensões (x,y) - formatação 2:
  //Obs: repare que essa formatação é mais restrita, pois precisa usar inteiros que tenham raíz exata!
  //dim3 dimBlock ((int) sqrt(N), (int) sqrt(N)); 
  //dim3 dimThreads((int) sqrt(N), (int) sqrt(N)); 

  // Imprime as posições acessadas pelo dimBlock e dimThreads
  printIndex<<< dimBlock, dimThreads>>>();

  // Execução do kernel matMult em GPU
  matMult<<< dimBlock, dimThreads>>>(dev_a, dev_b, dev_c);
  hipDeviceSynchronize();

  // Cópia do vetor (c) da GPU (Memória Global) para CPU
  hipMemcpy (c, dev_c, size, hipMemcpyDeviceToHost);

  // Impressão na tela dos valores dos vetores
  printf ("\t ### Valores após processamento em GPU ###\n");
  printf ("\t ### Matriz (a) ### \n");
  printMat(a);
  printf ("\t ### Matriz (b) ### \n");
  printMat(b);
  printf ("\t ### Matriz (c) ### \n");
  printMat(c);

  // Libera a Memória Global (GPU)
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  // Libera a Memória Global (CPU)
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);

  return 0;
}
